
#include <hip/hip_runtime.h>
# include <stdio.h>
# include <stdlib.h>
# include <math.h>
# include <time.h>
# define THREAD_PER_BLOCK 1024

unsigned long long N;
unsigned long long p,q;
unsigned long long e_final[5];



__global__
void find_e(unsigned long long phiN, unsigned long long d, unsigned long long *e, unsigned long long step) {
    unsigned long long j = blockIdx.x * blockDim.x + threadIdx.x;
    j = j * step + 1;
    // if(j < 0)
    //     printf ("hello\n");
    // printf("j = %d\n", j);
    if(j == 1 || j >= phiN)
        return;
    // printf("%d ~ %d\n", j, j+step);
    for(unsigned long long i = j; i < j + step; ++i) {
        // printf("d*j = %llu\n", d*j);
        if(d*i%phiN==1){
            *e = i;
            // printf("succeed\n");
        }    
    }
    return;
    
}




int main(int argc,char **argv)
{
	N=strtoul(argv[1],NULL,0);
	unsigned long long d=strtoul(argv[2],NULL,0);
	unsigned long long SQRTN = (unsigned long long)sqrt(N);
	unsigned long long *e;
	unsigned long long step = 100000;

    if(argc == 2) {
        printf("usage: ./CUDA <N> <d>");
        exit(1);
    }
//	cout << "before" << endl;

//	cout << "first\n";
	p = q = 0;
	for(unsigned long long i=1;2*i+1<=SQRTN;i++)
	{
		if(N%(2*i+1)==0)
		{
			p = 2*i+1;
			q = N/p;
		}
	}
//	cout << "fifth\n";
	unsigned long long phiN=(p-1)*(q-1);
    printf("phiN = %llu\n", phiN);
    printf("block num = %llu\n", phiN/step + 1);


    hipMalloc(&e, sizeof(unsigned long long)); 	
    e_final[0] = 0;
    hipMemcpy(e, e_final, sizeof(unsigned long long), hipMemcpyHostToDevice);

    // phiN = 100;
    // d = 3;
    find_e<<<(phiN/step + 1 + THREAD_PER_BLOCK - 1) / THREAD_PER_BLOCK, THREAD_PER_BLOCK>>>(phiN, d, e, step);

    hipMemcpy(e_final, e, sizeof(unsigned long long), hipMemcpyDeviceToHost);
	// for(unsigned long long j=2;j<phiN;j++)
	// {
	// 	if(d*j%phiN==1){
	// 		e = j;
	// 	}
	// }



//	cout << "sixth\n";
    printf("e = %llu\n", e_final[0]);

    // printf("finished\n");
	hipFree(e);
    return 0;
}
